
#include <hip/hip_runtime.h>
/* Copyright 2015 The TensorFlow Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
==============================================================================*/

#if GOOGLE_CUDA
#define EIGEN_USE_GPU
#include "third_party/eigen3/unsupported/Eigen/CXX11/Tensor"
#include "correlation_param.h"


__device__
int getThreadIdx_3D_3D(){
    int threadId =  (threadIdx.z * (blockDim.x * blockDim.y))
                 + (threadIdx.y * blockDim.x) + threadIdx.x;
    return threadId;
}

__global__ void CorrelationKernel(const float* a, const float*b,float* out, const int batch_size,const int num_rows, const int num_cols, const int depth,const int num_offsets, const int* g_offset_list)  {
    int one_d_size   = depth;
    int two_d_size   = one_d_size*num_cols;
    int three_d_size = two_d_size*num_rows;

    int num_offset_ints = 2*num_offsets;
    // Copy the offset list into shared memory to speed up access
    __shared__ int offset_list[CORRELATION_OPERATOR_LIST_SIZE];
    int mem_index = getThreadIdx_3D_3D();
    int total_block_size = blockDim.x * blockDim.y * blockDim.z;
    for( ; mem_index < num_offset_ints; mem_index+= total_block_size)
    {

       offset_list[mem_index] = g_offset_list[mem_index];
    }
    
    __syncthreads();

    int out1 = num_offsets;
    int out2 = num_cols * out1;
    int out3 = num_rows * out2;

    for (int i = blockIdx.z * blockDim.z + threadIdx.z; i < batch_size; i+= blockDim.z * gridDim.z) {
        for (int j = blockIdx.x * blockDim.x + threadIdx.x; j < num_rows; j += blockDim.x * gridDim.x) {
          for (int k = blockIdx.y*blockDim.y + threadIdx.y; k < num_cols; k += blockDim.y * gridDim.y) {
              for (int l =0; l < num_offsets; l++ ) {
                int j_offset = offset_list[2*l];
                int k_offset = offset_list[2*l+1];
                int min_j = 0;
                int max_j = num_rows;
                int min_k = 0;
                int max_k = num_cols;
                if(j_offset < 0){
                    min_j = -1*j_offset;
                }else{
                    max_j -= j_offset;
                }
                if(k_offset < 0){
                    min_k = -1*k_offset;
                }else{
                    max_k -= k_offset;
                }
                int a_root = three_d_size*i + two_d_size*j+one_d_size * k;
                int out_index = out3*i + out2*j+out1*k + l;
                out[out_index] =0 ;
                if( j >= min_j && j < max_j  && k >= min_k && k < max_k)
                {
                    int b_j = j+j_offset;
                    int b_k = k+k_offset;
                    int b_root = three_d_size*i + two_d_size*b_j+one_d_size * b_k;
                    for( int m = 0 ; m < depth; m++)
                    {
                         out[out_index]+= a[a_root+m]*b[b_root+m];
                    }
                    out[out_index]/= depth;
	    
                }
              }
            }
        }
      }


}


/// Take the tensor arrays (which are allocated on the GPU by TensorFlow's  context->allocate_output() call )
/// and spawn the correct number of CUDA threads on the GPU
void CorrelationKernelLauncher(const float* a, const float*b,float* out, const int batch_size,const int num_rows, const int num_cols, const int depth,const int num_offsets, const int* offset_list) {

  // Move the offset array to GPU, since this one was allocated by the std::vector on the CPU side 
  int *offset_array;
  cudaMalloc(&offset_array, 2*num_offsets * sizeof(int)); 
  cudaMemcpy(offset_array, offset_list, 2*num_offsets*sizeof(int), cudaMemcpyHostToDevice);
  // Address the image in blocks of size  1 (batch)x 16 (height)x 16 (width) x  num_channel (depth) 
  int mx = 16;
  int my = 16;
  int mz = 1;
  // Calculate how many blocks are needed to cover the whole image. 
  // This math is long-hand for int nz = ceil(batch_size/mz);
  int nz = (batch_size + mz -1)/mz;
  int ny = (num_cols + my - 1)/my;
  int nx = (num_rows + mx -1)/mx;

  // Use CUDA's dim3 structs to contain the block counts and block shapes
  dim3 blocks(nx,ny,nz);
  dim3 threadsPerBlock(mx,my,mz);
  // Call the CUDA Kernel
  CorrelationKernel<<<blocks, threadsPerBlock>>>(a, b, out,batch_size,num_rows,num_cols,depth,num_offsets,offset_array);
}

#endif
